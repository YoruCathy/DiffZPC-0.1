#include "hip/hip_runtime.h"
// TODO: how to init zs::Vector, Assign value?

// C++ includes
#include <hip/hip_runtime.h>
#include <iostream>
#include "zensim/cuda/memory/MemOps.hpp"
#include "zensim/cuda/Cuda.h"

#include "zensim/container/Vector.hpp"
#include "zensim/omp/execution/ExecutionPolicy.hpp"
using namespace std;

// autodiff include
#include <zensim/autodiff/reverse/var.hpp>
using namespace autodiff;

// The single-variable function for which derivatives are needed
var f(var x)
{
    return 1 + x + x*x + 1/x + log(x);
}

int main()
{
    using namespace zs;
    fmt::print("\n\n==========example-forward-single-variable-function===========\n\n");
    auto ompPol=omp_exec().profile(true);
    Vector<autodiff::var> val{1, memsrc_e::host};
    Vector<autodiff::var> val1{1, memsrc_e::host};
    for(int i=0; i!=val.size();++i){
        val[i]=2.0;
    }
    for(int i=0; i!=val1.size();++i){
        val1[i]=f(val[i]);
    }

    ompPol(zip(val,val1), [](auto &val, auto &val1) mutable {
        printf("val: %f\n", val);
        printf("val1: %f\n", val1);
        auto [tmp] = derivatives(val1, wrt(val));
        printf("uv: %f\n", tmp);
    });

//    var x = 2.0;   // the input variable x
//    var u = f(x);  // the output variable u
//
//    auto [ux] = derivatives(u, wrt(x)); // evaluate the derivative of u with respect to x
//
//    cout << "u = " << u << endl;  // print the evaluated output variable u
//    cout << "ux = " << ux << endl;  // print the evaluated derivative ux
}
