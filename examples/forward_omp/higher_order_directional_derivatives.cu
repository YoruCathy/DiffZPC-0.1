#include "hip/hip_runtime.h"
// TODO: need a Tuple

#include <hip/hip_runtime.h>
#include <iostream>
#include "zensim/cuda/memory/MemOps.hpp"
#include "zensim/cuda/Cuda.h"

#include "zensim/container/Vector.hpp"
#include "zensim/omp/execution/ExecutionPolicy.hpp"

// autodiff include
#include <zensim/autodiff/forward/real.hpp>
using namespace autodiff;

// The multi-variable function for which higher-order derivatives are needed (up to 4th order)
real4th f(real4th x, real4th y, real4th z)
{
    return sin(x) * cos(y) * exp(z);
}

int main()
{
//    using namespace zs;
//    fmt::print("\n\n==========example-forward-single-variable-function===========\n\n");
//    auto ompPol=omp_exec().profile(true);
//    Vector<autodiff::dual> val{1, memsrc_e::host};
//    Vector<autodiff::dual> val1{1, memsrc_e::host};
//    for(int i=0; i!=val.size();++i){
//        val[i]=10.0;
//    }
//    for(int i=0; i!=val.size();++i){
//        std::cout<<"val[i]"<<val[i]<<std::endl;
//    }
//
//    ompPol(zip(val,val1), [](auto &val, auto &val1) mutable {
//        printf("val: %f\n", val.val);
//        auto tmp = derivative(f, wrt(val), at(val));;
//        printf("val.grad: (%f)\n", val.grad);
//        printf("val.grad2: (%f)\n", tmp);
//    });

    real4th x = 1.0;
    real4th y = 2.0;
    real4th z = 3.0;

    auto dfdv = derivatives(f, along(1.0, 1.0, 2.0), at(x, y, z)); // the directional derivatives of f along direction v = (1, 1, 2) at (x, y, z) = (1, 2, 3)

    std::cout << "dfdv[0] = " << dfdv[0] << std::endl; // print the evaluated 0th order directional derivative of f along v (equivalent to f(x, y, z))
    std::cout << "dfdv[1] = " << dfdv[1] << std::endl; // print the evaluated 1st order directional derivative of f along v
    std::cout << "dfdv[2] = " << dfdv[2] << std::endl; // print the evaluated 2nd order directional derivative of f along v
    std::cout << "dfdv[3] = " << dfdv[3] << std::endl; // print the evaluated 3rd order directional derivative of f along v
    std::cout << "dfdv[4] = " << dfdv[4] << std::endl; // print the evaluated 4th order directional derivative of f along v
}

/*-------------------------------------------------------------------------------------------------
=== Note ===
---------------------------------------------------------------------------------------------------
This example would also work if dual was used instead of real. However, real
types are your best option for directional derivatives, as they were optimally
designed for this kind of derivatives.
-------------------------------------------------------------------------------------------------*/
