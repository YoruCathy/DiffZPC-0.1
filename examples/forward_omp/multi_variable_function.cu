#include "hip/hip_runtime.h"
// TODO: need a tuple

#include <hip/hip_runtime.h>
#include <iostream>
#include "zensim/cuda/memory/MemOps.hpp"
#include "zensim/cuda/Cuda.h"

#include "zensim/container/Vector.hpp"
#include "zensim/omp/execution/ExecutionPolicy.hpp"
#include "zensim/container/TileVector.hpp"
#include "zensim/types/Tuple.h"
#include "zensim/math/matrix/Matrix.hpp"

#include "zensim/autodiff/forward/dual.hpp"
using namespace autodiff;

// The multi-variable function for which derivatives are needed
constexpr dual f(dual x, dual y, dual z)
{
  return 1 + x + y + z + x*y + y*z + x*z + x*y*z + exp(x/y + y/z);
}

int main() {
  using namespace zs;
  fmt::print("\n\n==========example-forward-single-variable-function===========\n\n");
  auto ompPol=omp_exec().profile(true);
  //  Vector<float> val{3, memsrc_e::host};
  //  tuple<Vector<float>> vals;
  Vector<tuple<autodiff::dual, autodiff::dual, autodiff::dual>> val{1, memsrc_e::host};
  val[0] = {1.0, 2.0, 3.0};
  //



  //  for(int i=0; i!=val.size();++i){
  //    std::cout<<"val[i]"<<val[i]<<std::endl;
  //  }

  ompPol(Collapse{val.size()}, [val = proxy<execspace_e::openmp>(val)](int i) mutable {
    auto xyz = val[i];
    autodiff::dual x = get<0>(xyz);
    autodiff::dual y = get<1>(xyz);
    autodiff::dual z = get<2>(xyz);
    //    double dudx = derivative(f, wrt(x), at(x, y, z));
    double dudx = derivative(f, wrt(x), at(x, y, z));
    double dudy = derivative(f, wrt(y), at(x, y, z));
    double dudz = derivative(f, wrt(z), at(x, y, z));

    printf("x.grad: (%f)\n", dudx);
    printf("y.grad: (%f)\n", dudy);
    printf("z.grad: (%f)\n", dudz);
  });

  /* -----old example ------*/
  //    dual x = 1.0;
  //    dual y = 2.0;
  //    dual z = 3.0;
  //
  //    dual u = f(x, y, z);
  //
  //    double dudx = derivative(f, wrt(x), at(x, y, z));
  //    double dudy = derivative(f, wrt(y), at(x, y, z));
  //    double dudz = derivative(f, wrt(z), at(x, y, z));
  //
  //    std::cout << "u = " << u << std::endl;         // print the evaluated output u = f(x, y, z)
  //    std::cout << "du/dx = " << dudx << std::endl;  // print the evaluated derivative du/dx
  //    std::cout << "du/dy = " << dudy << std::endl;  // print the evaluated derivative du/dy
  //    std::cout << "du/dz = " << dudz << std::endl;  // print the evaluated derivative du/dz

  return 0;
}