#include "hip/hip_runtime.h"
// TODO: this does not work

//#include <hip/hip_runtime.h>
//#include <iostream>
//#include <complex>
//using namespace std;
//
//#include "zensim/autodiff/forward/dual.hpp"
//#include "zensim/cuda/memory/MemOps.hpp"
//#include "zensim/cuda/Cuda.h"
//#include "zensim/container/Vector.hpp"
//#include "zensim/omp/execution/ExecutionPolicy.hpp"
//
//using namespace autodiff;
//// Specialize isArithmetic for complex to make it compatible with dual
//namespace autodiff::detail {
//
//    template<typename T>
//    struct ArithmeticTraits<complex<T>> : ArithmeticTraits<T> {};
//
//} // autodiff::detail
//
//using cxdual = Dual<complex<double>, complex<double>>;
//
//// The single-variable function for which derivatives are needed
//cxdual f(cxdual x)
//{
//    return log(x) + 1.0 + x + x*x + 1/x;
//}
//
//int main() {
//  using namespace zs;
//  fmt::print("\n\n==========example-forward-single-variable-function-custom-scalar===========\n\n");
//  auto ompPol=omp_exec().profile(true);
//  Vector<cxdual> val{1, memsrc_e::host};
//  Vector<cxdual> val1{1, memsrc_e::host};
//  for(int i=0; i!=val.size();++i){
//    val[i]=10.0;
//  }
//  for(int i=0; i!=val.size();++i){
//    std::cout<<"val[i]"<<val[i]<<std::endl;
//  }
//
//  ompPol(zip(val,val1), [](auto &val, auto &val1) mutable {
////    printf("i:%f\n", i);
////    printf("val[i]: (%f)\n", val[i]);
//    printf("val: %f\n", val.val);
//    //    printf("val: %f\n", val.grad);
//    auto tmp = derivative(f, wrt(val), at(val));;
//    printf("val.grad: (%f)\n", val.grad);
//    printf("val.grad2: (%f)\n", tmp);
//  });
//
//
///* -----old example ------*/
////  autodiff::dual x = 2.0;                                 // the input variable x
////  autodiff::dual u = f(x);                                // the output variable u
////
////  double dudx = derivative(f, wrt(x), at(x));   // evaluate the derivative du/dx
////
////  std::cout << "u = " << u << std::endl;        // print the evaluated output u
////  std::cout << "du/dx = " << dudx << std::endl; // print the evaluated derivative du/dx
//
//  return 0;
//}

// C++ includes
#include <iostream>
#include <complex>
using namespace std;

// autodiff include
#include <zensim/autodiff/forward/dual.hpp>
using namespace autodiff;

// Specialize isArithmetic for complex to make it compatible with dual
namespace autodiff::detail {

    template<typename T>
    struct ArithmeticTraits<complex<T>> : ArithmeticTraits<T> {};

} // autodiff::detail

using cxdual = Dual<complex<double>, complex<double>>;

// The single-variable function for which derivatives are needed
cxdual f(cxdual x)
{
    return 1 + x + x*x + 1/x + log(x);
}

int main()
{
    cxdual x = 2.0;   // the input variable x
    cxdual u = f(x);  // the output variable u

    cxdual dudx = derivative(f, wrt(x), at(x));  // evaluate the derivative du/dx

    cout << "u = " << u << endl;         // print the evaluated output u
    cout << "du/dx = " << dudx << endl;  // print the evaluated derivative du/dx
}
