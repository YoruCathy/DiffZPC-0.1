#include "hip/hip_runtime.h"
// TODO: fit the data structure

#include <hip/hip_runtime.h>
#include <iostream>
#include "zensim/cuda/memory/MemOps.hpp"
#include "zensim/cuda/Cuda.h"

#include "zensim/container/Vector.hpp"
#include "zensim/omp/execution/ExecutionPolicy.hpp"

#include "zensim/autodiff/forward/dual.hpp"
using namespace autodiff;

// A type defining parameters for a function of interest
struct Params
{
  dual a;
  dual b;
  dual c;
};

// The function that depends on parameters for which derivatives are needed
dual f(dual x, const Params& params)
{
  return params.a * sin(x) + params.b * cos(x) + params.c * sin(x)*cos(x);
}

int main()
{
  using namespace zs;
  fmt::print("\n\n==========example-forward-single-variable-function===========\n\n");
  auto ompPol=omp_exec().profile(true);
  Vector<autodiff::dual> val{1, memsrc_e::host};
  Vector<Params> param{1, memsrc_e::host};
  //  params.a = 1.0;  // the parameter a of type dual, not double!
  //  params.b = 2.0;  // the parameter b of type dual, not double!
  //  params.c = 3.0;  // the parameter c of type dual, not double!

  val[0] = 0.5;
  param[0].a = 1.0;
  param[0].b = 2.0;
  param[0].c = 3.0;

  ompPol(zip(val,param), [](auto &val, auto &param) mutable {
    dual x = val;
    double dudx = derivative(f, wrt(x), at(x, param));
    double duda = derivative(f, wrt(param.a), at(x, param)); // evaluate the derivative du/da
    double dudb = derivative(f, wrt(param.b), at(x, param)); // evaluate the derivative du/db
    double dudc = derivative(f, wrt(param.c), at(x, param)); // evaluate the derivative du/dc
    printf("val.grad: (%f)\n", dudx);
    printf("val.grad: (%f)\n", duda);
    printf("val.grad: (%f)\n", dudb);
    printf("val.grad: (%f)\n", dudc);
  });
  return 0;

  /* original example*/
  //    Params params;   // initialize the parameter variables
  //    params.a = 1.0;  // the parameter a of type dual, not double!
  //    params.b = 2.0;  // the parameter b of type dual, not double!
  //    params.c = 3.0;  // the parameter c of type dual, not double!
  //
  //    dual x = 0.5;  // the input variable x
  //
  //    dual u = f(x, params);  // the output variable u
  //
  //    double dudx = derivative(f, wrt(x), at(x, params));        // evaluate the derivative du/dx
  //    double duda = derivative(f, wrt(params.a), at(x, params)); // evaluate the derivative du/da
  //    double dudb = derivative(f, wrt(params.b), at(x, params)); // evaluate the derivative du/db
  //    double dudc = derivative(f, wrt(params.c), at(x, params)); // evaluate the derivative du/dc
  //
  //    std::cout << "u = " << u << std::endl;         // print the evaluated output u
  //    std::cout << "du/dx = " << dudx << std::endl;  // print the evaluated derivative du/dx
  //    std::cout << "du/da = " << duda << std::endl;  // print the evaluated derivative du/da
  //    std::cout << "du/db = " << dudb << std::endl;  // print the evaluated derivative du/db
  //    std::cout << "du/dc = " << dudc << std::endl;  // print the evaluated derivative du/dc
}

/*-------------------------------------------------------------------------------------------------
=== Note ===
---------------------------------------------------------------------------------------------------
This example would also work if real was used instead of dual. Should you
need higher-order cross derivatives, however, e.g.,:
    double d2udxda = derivative(f, wrt(x, params.a), at(x, params));
then higher-order dual types are the right choicesince real types are
optimally designed for higher-order directional derivatives.
-------------------------------------------------------------------------------------------------*/