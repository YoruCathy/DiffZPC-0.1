#include <hip/hip_runtime.h>
#include <iostream>
#include "zensim/cuda/memory/MemOps.hpp"
#include "zensim/cuda/Cuda.h"

#include "zensim/container/Vector.hpp"
#include "zensim/omp/execution/ExecutionPolicy.hpp"

#include "zensim/autodiff/forward/dual.hpp"
//using namespace autodiff;

constexpr autodiff::dual f(autodiff::dual x)
{
autodiff::dual tmp = 1 + x + x*x;
static_assert(autodiff::detail::isExpr<RM_CVREF_T(tmp)>, "this is indeed an expr");
return tmp;
}

int main() {
    using namespace zs;
    fmt::print("\n\n==========example-forward-single-variable-function===========\n\n");
    auto ompPol=omp_exec().profile(true);
    Vector<autodiff::dual> val{1, memsrc_e::host};
    Vector<autodiff::dual> val1{1, memsrc_e::host};
    for(int i=0; i!=val.size();++i){
        val[i]=10.0;
    }
    for(int i=0; i!=val.size();++i){
        std::cout<<"val[i]"<<val[i]<<std::endl;
    }

    ompPol(zip(val,val1), [](auto &val, auto &val1) mutable {
        printf("val: %f\n", val.val);
        auto tmp = derivative(f, wrt(val), at(val));;
        printf("val.grad: (%f)\n", val.grad);
        printf("val.grad2: (%f)\n", tmp);
    });

/* -----old example ------*/
//  autodiff::dual x = 2.0;                                 // the input variable x
//  autodiff::dual u = f(x);                                // the output variable u
//
//  double dudx = derivative(f, wrt(x), at(x));   // evaluate the derivative du/dx
//
//  std::cout << "u = " << u << std::endl;        // print the evaluated output u
//  std::cout << "du/dx = " << dudx << std::endl; // print the evaluated derivative du/dx

    return 0;
}