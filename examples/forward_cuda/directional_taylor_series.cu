#include "hip/hip_runtime.h"
// TODO: need a Tuple

#include <hip/hip_runtime.h>
#include <iostream>
#include "zensim/cuda/memory/MemOps.hpp"
#include "zensim/cuda/Cuda.h"

#include "zensim/container/Vector.hpp"
#include "zensim/omp/execution/ExecutionPolicy.hpp"

// autodiff include
#include <zensim/autodiff/forward/real.hpp>
using namespace autodiff;

// The scalar function for which a 4th order directional Taylor series will be computed.
real4th f(const real4th& x, const real4th& y, const real4th& z)
{
    return sin(x * y) * cos(x * z) * exp(z);
}

int main()
{
//    using namespace zs;
//    fmt::print("\n\n==========example-forward-single-variable-function===========\n\n");
//    auto ompPol=omp_exec().profile(true);
//    Vector<autodiff::dual> val{1, memsrc_e::host};
//    Vector<autodiff::dual> val1{1, memsrc_e::host};
//    for(int i=0; i!=val.size();++i){
//        val[i]=10.0;
//    }
//    for(int i=0; i!=val.size();++i){
//        std::cout<<"val[i]"<<val[i]<<std::endl;
//    }
//
//    ompPol(zip(val,val1), [](auto &val, auto &val1) mutable {
//        printf("val: %f\n", val.val);
//        auto tmp = derivative(f, wrt(val), at(val));;
//        printf("val.grad: (%f)\n", val.grad);
//        printf("val.grad2: (%f)\n", tmp);
//    });


    real4th x = 1.0;                                       // the input vector x
    real4th y = 2.0;                                       // the input vector y
    real4th z = 3.0;                                       // the input vector z

    auto g = taylorseries(f, along(1, 1, 2), at(x, y, z)); // the function g(t) as a 4th order Taylor approximation of f(x + t, y + t, z + 2t)

    double t = 0.1;                                        // the step length used to evaluate g(t), the Taylor approximation of f(x + t, y + t, z + 2t)

    real4th u = f(x + t, y + t, z + 2*t);                  // the exact value of f(x + t, y + t, z + 2t)

    double utaylor = g(t);                                 // the 4th order Taylor estimate of f(x + t, y + t, z + 2t)

    std::cout << std::fixed;
    std::cout << "Comparison between exact evaluation and 4th order Taylor estimate of f(x + t, y + t, z + 2t):" << std::endl;
    std::cout << "u(exact)  = " << u << std::endl;
    std::cout << "u(taylor) = " << utaylor << std::endl;
}

/*-------------------------------------------------------------------------------------------------
=== Output ===
---------------------------------------------------------------------------------------------------
Comparison between exact evaluation and 4th order Taylor estimate of f(x + t, y + t, z + 2t):
u(exact)  = -16.847071
u(taylor) = -16.793986
-------------------------------------------------------------------------------------------------*/
