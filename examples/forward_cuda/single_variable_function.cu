#include <hip/hip_runtime.h>
#include <iostream>
#include "zensim/cuda/memory/MemOps.hpp"
#include "zensim/cuda/Cuda.h"

#include "zensim/container/Vector.hpp"
#include "zensim/omp/execution/ExecutionPolicy.hpp"
#include "zensim/cuda/execution/ExecutionPolicy.cuh"

#include "zensim/autodiff/forward/dual.hpp"
//using namespace autodiff;

constexpr autodiff::dual f(autodiff::dual x)
{
//  autodiff::dual tmp = 1 + x + x*x;
//autodiff::dual tmp = 1;
//tmp += std::move(x);
//autodiff::dual tmp = autodiff::detail::AddExpr<autodiff::detail::PreventExprRef<int>, autodiff::detail::PreventExprRef<autodiff::dual>>{ 1, x };
//autodiff::dual tmp = autodiff::detail::BinaryExpr<autodiff::detail::AddOp, int, autodiff::dual>{1, x};
#if 0
    autodiff::dual tmp{};// = autodiff::detail::BinaryExpr<autodiff::detail::AddOp, int, int>{1, 2};
    assign(tmp, autodiff::detail::BinaryExpr<autodiff::detail::AddOp, int, autodiff::dual>{1, x});
#elif 1
  auto tmp = autodiff::dual::def(x + 1 + x * x);
#else
  autodiff::dual tmp{};
  assign(tmp, autodiff::detail::BinaryExpr<autodiff::detail::AddOp, int, int>{1, 2});
#endif
  //  static_assert(autodiff::detail::isExpr<RM_CVREF_T(tmp)>, "this is indeed an expr");
  return tmp;
}

int main() {
  using namespace zs;
  fmt::print("\n\n==========example-forward-single-variable-function===========\n\n");
  auto cudaPol= cuda_exec().profile(true);
  Vector<autodiff::dual> val{1, memsrc_e::device, 0};

  cudaPol(enumerate(val), [] __device__(auto id, auto &v) mutable {
    v = 10;
    printf("v: %f", (float)v);
  });
  cudaPol(Collapse{val.size()}, [val = proxy<execspace_e::cuda>(val)] __device__(auto id) mutable {
    autodiff::dual v_ = val[id];
    printf("v_: %f", (float)v_);
    //    autodiff::dual* v = &v_;
    double dudx = derivative(f, wrt(v_), at(v_));
    printf("val.grad: (%f)\n", dudx);
    auto u = f(v_);
    printf("u: (%f)\n", (float)u);
  });
  /* -----old example ------*/
  //  autodiff::dual x = 2.0;                                 // the input variable x
  //  autodiff::dual u = f(x);                                // the output variable u
  //
  //  double dudx = derivative(f, wrt(x), at(x));   // evaluate the derivative du/dx
  //
  //  std::cout << "u = " << u << std::endl;        // print the evaluated output u
  //  std::cout << "du/dx = " << dudx << std::endl; // print the evaluated derivative du/dx

  return 0;
}