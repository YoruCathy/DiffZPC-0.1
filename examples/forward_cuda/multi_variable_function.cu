#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "zensim/cuda/memory/MemOps.hpp"
#include "zensim/cuda/Cuda.h"

#include "zensim/container/Vector.hpp"
#include "zensim/omp/execution/ExecutionPolicy.hpp"
#include "zensim/cuda/execution/ExecutionPolicy.cuh"

#include "zensim/autodiff/forward/dual.hpp"
//using namespace autodiff;

constexpr autodiff::dual f(autodiff::dual x, autodiff::dual y, autodiff::dual z)
{
  auto tmp = autodiff::dual::def(1 + x + y + z + x*y + y*z + x*z + x*y*z + exp(x/y + y/z));
  return tmp;
}

int main() {
  using namespace zs;
  //  fmt::print("\n\n==========example-forward-single-variable-function===========\n\n");
  auto cudaPol=cuda_exec().profile(true);
  //  Vector<float> val{3, memsrc_e::host};
  //  tuple<Vector<float>> vals;
  Vector<tuple<autodiff::dual, autodiff::dual, autodiff::dual>> val{1, memsrc_e::device, 0};
  //  val[0] = {1.0, 2.0, 3.0};
  //

  cudaPol(enumerate(val), [] __device__(auto id, auto &v) mutable {
    v = {1.0, 2.0, 3.0};
    //    printf("v: %f", (float)v);
  });

  //  for(int i=0; i!=val.size();++i){
  //    std::cout<<"val[i]"<<val[i]<<std::endl;
  //  }

  cudaPol(Collapse{val.size()}, [val = proxy<execspace_e::cuda>(val)]__device__(int i) mutable {
    auto xyz = val[i];
    autodiff::dual x = get<0>(xyz);
    autodiff::dual y = get<1>(xyz);
    autodiff::dual z = get<2>(xyz);
    //    double dudx = derivative(f, wrt(x), at(x, y, z));
    double dudx = derivative(f, wrt(x), at(x, y, z));
    double dudy = derivative(f, wrt(y), at(x, y, z));
    double dudz = derivative(f, wrt(z), at(x, y, z));

    printf("x.grad: (%f)\n", dudx);
    printf("y.grad: (%f)\n", dudy);
    printf("z.grad: (%f)\n", dudz);
  });
  /* -----old example ------*/
  //  autodiff::dual x = 2.0;                                 // the input variable x
  //  autodiff::dual u = f(x);                                // the output variable u
  //
  //  double dudx = derivative(f, wrt(x), at(x));   // evaluate the derivative du/dx
  //
  //  std::cout << "u = " << u << std::endl;        // print the evaluated output u
  //  std::cout << "du/dx = " << dudx << std::endl; // print the evaluated derivative du/dx

  return 0;
}